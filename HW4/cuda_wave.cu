/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265
#define THREAD 216

void check_param(void);
__global__ void update(int tpoints, int nsteps, float *values);
void printfinal(void);

int nsteps,                     /* number of time steps */
    tpoints,                    /* total points along string */
    rcode;                      /* generic return code */
float  values[MAXPOINTS+2];     /* values at time t */
       //oldval[MAXPOINTS+2],   /* values at time (t-dt) */
       //newval[MAXPOINTS+2];   /* values at time (t+dt) */

/**********************************************************************
 *      Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n",
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}
/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void update(int tpoints, int nsteps, float *values)
{
        int i;
        int j = blockIdx.x * blockDim.x + threadIdx.x;
        float x, fac, tmp;
        __shared__ float value, oldval, newval;
        /* Calculate initial values based on sine curve */
        fac = 2.0 * PI;
        tmp = tpoints - 1;
        x = (float)(j-1)/tmp;
        value = sin (fac * x);
        oldval = value;
        //float dtime, c, dx, tau;
        float sqtau;
        //dtime = 0.3;
        //c = 1.0;
        //dx = 1.0;
        //tau = (c * dtime / dx);
        sqtau = 0.3 * 0.3;
        /* Update values for each time step */
        if ((j == 1) || (j == tpoints))
                value = 0.0;
        else {
                for (i = 1; i<= nsteps; i++) {
                        //newval = 1.82 * value - oldval;
                        newval = (2.0 * (1.0 - sqtau) * value) - oldval;
                        oldval = value;
                        value = newval;
                }
        }
        values[j] = value;
}
/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}
/**********************************************************************
 *      Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
        sscanf(argv[1],"%d",&tpoints);
        sscanf(argv[2],"%d",&nsteps);
        check_param();

        float *cuda_values;
        int size = (tpoints+1) * sizeof(float);
        hipMalloc(&cuda_values, size);

        printf("Initializing points on the line...\n");
        printf("Updating all points for all time steps...\n");
        update<<<(tpoints+THREAD)/THREAD, THREAD>>>(tpoints, nsteps, cuda_values);
        hipMemcpy(values, cuda_values, size, hipMemcpyDeviceToHost);
        printf("Printing final results...\n");
        printfinal();
        printf("\nDone.\n\n");

        hipFree(cuda_values);

        return 0;
}
