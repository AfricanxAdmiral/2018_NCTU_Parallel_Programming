
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include <complex>
#include <cstring>
#include <iostream>
#include <fstream>
using namespace std;

const int N = (1 << 30);

__global__ void multiply(int n, int m, char x[], char y[], int ans[]) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if( i < n && j < m){
        int a = n-1-i, b = m-1-j;
        atomicAdd(&ans[a+b], (x[i]-48) * (y[j]-48));
        //printf("x[%d] : %d * y[%d] : %d = %d\n", i, x[i]-48, j, y[j]-48, ans[a+b]);
    }
}

int main(int argc,char *argv[]) {
    char *s = new char[N];
    char *t = new char[N];

    int n, m, l;

    FILE* fin = fopen(argv[1],"r");
    fscanf(fin,"%s",s);
    fclose(fin);

    fin = fopen(argv[2],"r");
    fscanf(fin,"%s",t);
    fclose(fin);

    n = strlen(s);
    m = strlen(t);
    l = n + m + 1;

    int *ans = new int[l];

    char* cuda_s;
    char* cuda_t;
    int* cuda_ans;
    hipMalloc(&cuda_s, n * sizeof(char));
    hipMalloc(&cuda_t, m * sizeof(char));
    hipMalloc(&cuda_ans, l * sizeof(int));

    hipMemcpy(cuda_s, s, n*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(cuda_t, t, m*sizeof(char), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(n+16 / threadsPerBlock.x, m+16 / threadsPerBlock.y);
    multiply<<<numBlocks, threadsPerBlock>>>(n, m, cuda_s, cuda_t, cuda_ans);

    hipMemcpy(ans, cuda_ans, l*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < l; ++i) {
        ans[i + 1] += ans[i] / 10;
        ans[i] %= 10;
    }

    int p = l;

    for (; p && !ans[p]; --p);
    for (; ~p; putchar(ans[p--] + '0'));
    puts("");

    return 0;
}
